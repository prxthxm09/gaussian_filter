#include "hip/hip_runtime.h"
%%writefile gaussian_filter.cu
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#define MASK_WIDTH 5
#define TILE_WIDTH 16
#define BLOCK_WIDTH (TILE_WIDTH + MASK_WIDTH - 1)
#define CHANNELS 3

__constant__ float mask[MASK_WIDTH * MASK_WIDTH];

__global__ void gaussianFilter(unsigned char *input, unsigned char *output, int width, int height, int channels) {
    __shared__ float tile[BLOCK_WIDTH][BLOCK_WIDTH][CHANNELS];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row_o = blockIdx.y * TILE_WIDTH + ty;
    int col_o = blockIdx.x * TILE_WIDTH + tx;
    int row_i = row_o - MASK_WIDTH / 2;
    int col_i = col_o - MASK_WIDTH / 2;

    for (int c = 0; c < channels; ++c) {
        if ((row_i >= 0) && (row_i < height) && (col_i >= 0) && (col_i < width)) {
            tile[ty][tx][c] = input[(row_i * width + col_i) * channels + c];
        } else {
            tile[ty][tx][c] = 0.0f;
        }
    }

    __syncthreads();

    float p_value[CHANNELS] = {0.0f};

    if (ty < TILE_WIDTH && tx < TILE_WIDTH) {
        for (int i = 0; i < MASK_WIDTH; ++i) {
            for (int j = 0; j < MASK_WIDTH; ++j) {
                for (int c = 0; c < channels; ++c) {
                    p_value[c] += mask[i * MASK_WIDTH + j] * tile[i + ty][j + tx][c];
                }
            }
        }
        for (int c = 0; c < channels; ++c) {
            if (row_o < height && col_o < width) {
                output[(row_o * width + col_o) * channels + c] = (unsigned char) p_value[c];
            }
        }
    }
}

void initGaussianMask(float sigma) {
    float kernel[MASK_WIDTH * MASK_WIDTH];
    float sum = 0.0f;
    int half_width = MASK_WIDTH / 2;

    for (int i = -half_width; i <= half_width; ++i) {
        for (int j = -half_width; j <= half_width; ++j) {
            kernel[(i + half_width) * MASK_WIDTH + (j + half_width)] = expf(-(i * i + j * j) / (2.0f * sigma * sigma));
            sum += kernel[(i + half_width) * MASK_WIDTH + (j + half_width)];
        }
    }

    for (int i = 0; i < MASK_WIDTH * MASK_WIDTH; ++i) {
        kernel[i] /= sum;
    }

    hipMemcpyToSymbol(HIP_SYMBOL(mask), kernel, MASK_WIDTH * MASK_WIDTH * sizeof(float));
}

int main(int argc, char **argv) {
    if (argc != 3) {
        printf("Usage: %s <input_image> <output_image>\n", argv[0]);
        return 1;
    }

    const char *input_filename = argv[1];
    const char *output_filename = argv[2];

    int width, height, channels;
    unsigned char *input_image = stbi_load(input_filename, &width, &height, &channels, CHANNELS);
    if (input_image == NULL) {
        printf("Error loading the image: %s\n", input_filename);
        return 1;
    }

    size_t image_size = width * height * channels * sizeof(unsigned char);
    unsigned char *output_image = (unsigned char*)malloc(image_size);

    unsigned char *d_input, *d_output;

    hipMalloc(&d_input, image_size);
    hipMalloc(&d_output, image_size);

    hipMemcpy(d_input, input_image, image_size, hipMemcpyHostToDevice);

   
    initGaussianMask(1.0f);

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid((width - 1) / TILE_WIDTH + 1, (height - 1) / TILE_WIDTH + 1);

    gaussianFilter<<<dimGrid, dimBlock>>>(d_input, d_output, width, height, channels);

    hipMemcpy(output_image, d_output, image_size, hipMemcpyDeviceToHost);

    stbi_write_png(output_filename, width, height, channels, output_image, width * channels);

    hipFree(d_input);
    hipFree(d_output);
    stbi_image_free(input_image);
    free(output_image);

    return 0;
}
